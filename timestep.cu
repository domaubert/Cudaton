#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cutil.h>
#include "params.h"
#include "common.h"
#include "bnd.h"
#include "GPU.h"
#include "Boundary.h"
#include "cosmo.h"
#include "Allocation.h"
#include "Io.h"
#include "Explicit.h"
#include "Atomic.h"
#ifdef WMPI
#include "communication.h"
#include "Interface.h"
#endif



//**********************************************************
//**********************************************************

extern "C" int Mainloop(int rank, int *pos, int *neigh, int ic_rank);

//**********************************************************
//**********************************************************


#define CUERR() //printf("\n %s on %d \n",hipGetErrorString(hipGetLastError()),ic_rank)

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef min
#define min( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define NCELLS3 (NCELLX+NBOUND2)*(NCELLY+NBOUND2)*(NCELLZ+NBOUND2)

#define N_INT 2048
#define A_INT_MAX 0.166667


//**********************************************************
//**********************************************************


int Mainloop(int rank, int *pos, int *neigh, int ic_rank)
{

  if(rank==0) printf("Mainloop entered by proc %d\n",rank);

  float tnext;


  dim3 blockion(NCELLX);           // USED BY IONISATION
  dim3 gridion(NCELLY,NCELLZ);

  dim3 bcool(BLOCKCOOL);           // USED BY COOLING
  dim3 gcool(GRIDCOOLX,GRIDCOOLY);
  
  dim3 blocksimple(NCELLX);        // USED BY ADVECTION THREADS
  dim3 gridsimple(NCELLY,NCELLZ);


#ifdef SDISCRETE
  int nthreadsource=min(nsource,128);
  dim3 gridsource((int)(round((float)(nsource)/float(nthreadsource))));
  dim3 blocksource(nthreadsource);
#endif

#ifndef WMPI

  dim3 blockboundx(NCELLY);
  dim3 gridboundx(NCELLZ);

  dim3 blockboundy(NCELLX);
  dim3 gridboundy(NCELLZ);

  dim3 blockboundz(NCELLX);
  dim3 gridboundz(NCELLY);

for (int igrp=0;igrp<NGRP;igrp++)
	{
  if(boundary==0) // transmissive boundary conditions
    {
      cusetboundarytrans_xp<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundarytrans_yp<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundarytrans_zp<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundarytrans_xm<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundarytrans_ym<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundarytrans_zm<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
    }
  else if(boundary==1) // reflexive boundary conditions
    {
      cusetboundaryref_zp<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_zm<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_yp<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_ym<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_xp<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_xm<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
    }
  else if(boundary==2) // Periodic boundary conditions
    {
      cusetboundaryper_xp<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_yp<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_zp<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_xm<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_ym<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_zm<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
    }
  else if(boundary==3) // Mixed boundary conditions
    {
      cusetboundarytrans_xp<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_yp  <<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_zp  <<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundarytrans_xm<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_ym  <<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_zm  <<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
    }
  }
#else
  dim3 blockboundx(NCELLY);
  dim3 gridboundx(NCELLZ);

  dim3 blockboundy(NCELLX);
  dim3 gridboundy(NCELLZ);

  dim3 blockboundz(NCELLX);
  dim3 gridboundz(NCELLY);


  if(neigh[5]!=rank)  
    {  
      exchange_zp(cuegy, cuflx, cuegy_new, buff, neigh, pos[2]%2);
      exchange_zm(cuegy, cuflx, cuegy_new, buff, neigh, pos[2]%2);
    }
  else
    {
      cusetboundaryper_zp<<<gridboundz,blockboundz>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      cusetboundaryper_zm<<<gridboundz,blockboundz>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
    }
  
  if(neigh[3]!=rank)
    {
      exchange_yp(cuegy, cuflx, cuegy_new, buff, neigh, pos[1]%2);
      exchange_ym(cuegy, cuflx, cuegy_new, buff, neigh, pos[1]%2);
    }
  else
    {
      cusetboundaryper_yp<<<gridboundy,blockboundy>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      cusetboundaryper_ym<<<gridboundy,blockboundy>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
    }

  if(neigh[1]!=rank)
    {
      exchange_xp(cuegy, cuflx, cuegy_new, buff, neigh, pos[0]%2);
      exchange_xm(cuegy, cuflx, cuegy_new, buff, neigh, pos[0]%2);
    }
  else
    {
      cusetboundaryper_xp<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      cusetboundaryper_xm<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
    }


  if(boundary==0)
    {
      if(pos[0]==0) cusetboundarytrans_xm<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      if(pos[1]==0) cusetboundarytrans_ym<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      if(pos[2]==0) cusetboundarytrans_zm<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);

      if(pos[0]==(NGPUX-1)) cusetboundarytrans_xp<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      if(pos[1]==(NGPUY-1)) cusetboundarytrans_yp<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      if(pos[2]==(NGPUZ-1)) cusetboundarytrans_zp<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);

    }

#endif


#ifndef COSMO
  dt=courantnumber*dx/3./c;
  if(rank==0) printf("dx=%e cfl=%e dt=%e\n",dx,courantnumber,dt);
  tnext=t;//+ndumps*dt;
#else
  aexp=astart;
#ifndef FLAT_COSMO
  t=a2tgen(aexp,omegam,omegav,Hubble0);// Hubble0 in sec-1
#else
  t=a2t(aexp,omegav,Hubble0);// Hubble0 in sec-1
#endif
  
  tnext=t;
  float tstart=t;
  if(rank==0) printf("aexp= %f tstart=%f tmax=%f\n",aexp,t/unit_time,tmax/unit_time);

#ifndef FLAT_COSMO
  if(rank==0) printf("Building Expansion factor table");

  float da=(A_INT_MAX-aexp)/N_INT;
  float a_int[N_INT],t_int[N_INT];
  for(int i_int=0;i_int<N_INT;i_int++)
    {
      a_int[i]=aexp+i_int*da;
      t_int[i]=a2tgen(a_int[i],omegam,omegav,Hubble0); // Hubble0 in sec-1
    }

  int n_int=0;

#endif


#endif
  
  // some variables for field update
  int changefield=0;
  int forcedump;
  int ifield=0; // 1 because tfield stores the NEXT field
  float tfield;
  if(fieldlist){
    while(t>=tlist[ifield])
      {
	ifield++;
      }
    tfield=tlist[ifield];
    if(rank==0) printf("ICs (tstart=%f) between field #%d (t=%f) and field #%d (t=%f)\n",t/unit_time,ifield-1,tlist[ifield-1]/unit_time,ifield,tlist[ifield]/unit_time);
    if(rank==0) printf("starting with NEXT field #%d @ tfield =%f with tstart=%f\n",ifield,tlist[ifield]/unit_time,t/unit_time);// -1 because tfield stores the NEXT field
  }

  // log file
  FILE *logfile;
  if(rank==0) logfile=fopen("log.out","w");

#ifdef TIMINGS
  FILE *timefile;
  if(rank==0)
    {
      timefile=fopen("time.out","w");
    }
#endif

  //float ft=1./powf(2.,20);
  float ft=1.;
#ifdef COSMO
  float factfesc=1.;
#endif

float *factgrp;
factgrp=(float*)malloc(NGRP*sizeof(float));
FACTGRP;

  unsigned int timer;
  float q0=0.,q1=0.,q3;
#ifdef TIMINGS
  float q4,q7,q8,q9,q10,q11;
  double time_old,time_new;
#endif  
  if(rank==0)
    {
      cutCreateTimer(&timer);
      cutStartTimer(timer);

    }

  
  // MAIN LOOP STARTS HERE ======================================================>>>>
  // ============================================================================>>>>
  // ============================================================================>>>>
  // ============================================================================>>>>
  // ============================================================================>>>>
  // ============================================================================>>>>

  hipDeviceSynchronize();
#ifdef WMPI	  
  mpisynch();
#endif
  
  cuDumpResults(0,t,aexp,0);

  while(t<=tmax)
    {  
      

      hipDeviceSynchronize();
#ifdef WMPI	  
      get_elapsed(&time_old);
      mpisynch();
#endif
      if(rank==0)
	{
	  q3=q1-q0;
	  q0=cutGetTimerValue(timer);
	}
      

#ifndef COSMO
      dt=courantnumber*dx/3./c*ft;
      if(((nstep%ndisp)==0)&&(rank==0))
	{
	  printf(" ------------------ \n");
	  printf(" Step= %d Time= %f dt=%f tnext=%f cgpu (msec)=%f\n",nstep,t/unit_time,dt/unit_time,tnext/unit_time,q3);
	  printf(" ------------------ \n");
	}
#else
      dt=courantnumber*dx/3./c*ft;

      if(((nstep%ndisp)==0)&&(rank==0))
	{
	  printf(" ------------------------------\n");
	  printf(" Step= %d Time= %f Elapsed= %f dt= %f aexp=%f z=%f fesc=%f clump= %f Next tfield=%f cgpu=%f\n",nstep,t/unit_time,(t-tstart)/unit_time,dt/unit_time,aexp,1./aexp-1.,factfesc*fesc,clump,tfield/unit_time,q3);
	  printf(" ----------------------------- \n");
	  fprintf(logfile,"%d %f %f %f %f %f %f %f\n",nstep,t/unit_time,(t-tstart)/unit_time,dt/unit_time,aexp,1./aexp-1.,tfield/unit_time,q3);
	}
#endif
      

      if(fieldlist)
	{
	  // we must not go further than the next field
	  if(dt>=tfield-t)
	    {
#ifdef WMPI
	      if(rank==0) printf("last timestep with field #%d : next field= %f t=%f t+dt=%f\n",ifield,tfield/unit_time,t/unit_time,(t+dt)/unit_time);

	      if(((tfield-t)/unit_time)==0.)
		{
		  if(rank==0) printf("WARNING FIELD DT=O -> switch immediatly to next field\n"); 
		  cuGetField(ifield,ic_rank);
		  changefield=0;
		  ifield++;
		  tfield=tlist[ifield];
		  ft=1./powf(2.,20);
		}
	      else
		{
		  changefield=1;
		  dt=tfield-t;
		  if(rank==0) printf("dt set to %f\n",dt/unit_time);
		}
#else
	      if(rank==0) printf("last timestep with field #%d : next field= %f t=%f t+dt=%f\n",ifield,tfield/unit_time,t/unit_time,(t+dt)/unit_time);

	      if(((tfield-t)/unit_time)==0.)
		{
		  if(rank==0) printf("WARNING FIELD DT=O -> switch immediatly to next field\n"); 
		  cuGetField(ifield,ic_rank);
		  changefield=0;
		  ifield++;
		  tfield=tlist[ifield];
		  ft=1./powf(2.,20);
		}
	      else
		{
		  changefield=1;
		  dt=tfield-t;
		  if(rank==0) printf("dt set to %f\n",dt/unit_time);
		}
#endif
	    }
	}

      //================================== UNSPLIT 3D SCHEME=============================


	for (int igrp=0;igrp<NGRP;igrp++)
		{
		#ifdef COSMO
		      cuComputeELF<<<gridsimple,blocksimple>>>(cuegy+igrp*NCELLS3, cuflx+igrp*NCELLS3*3, cusrc0, cuegy_new+igrp*NCELLS3, c, dx, dt, nstep,aexp);
		#else
		      cuComputeELF<<<gridsimple,blocksimple>>>(cuegy+igrp*NCELLS3, cuflx+igrp*NCELLS3*3, cusrc0, cuegy_new+igrp*NCELLS3, c, dx, dt, nstep,1.);
		#endif
		
		      hipDeviceSynchronize();
		      CUERR();
			      if(verbose) puts("Hyperbolic Egy ok");
		
		#ifdef COSMO
		      cuComputeF_TOTAL_LF<<<gridsimple,blocksimple>>>(cuflx+igrp*NCELLS3*3,cudedd,cusrc0,cuflx_new+igrp*NCELLS3*3,c,dx,dt,nstep,cuegy+igrp*NCELLS3, aexp);
		#else
		      cuComputeF_TOTAL_LF<<<gridsimple,blocksimple>>>(cuflx+igrp*NCELLS3*3,cudedd,cusrc0,cuflx_new+igrp*NCELLS3*3,c,dx,dt,nstep,cuegy+igrp*NCELLS3,1.);
		#endif
		      hipDeviceSynchronize();
		      CUERR();
	
		#ifdef SDISCRETE
		#ifdef COSMO
		      if(kf!=0.) factfesc=exp(kf-powf(aexp/a0,af));
		      cuAddSource<<<gridsource,blocksource>>>(cuegy_new+igrp*NCELLS3,cuflx_new+igrp*NCELLS3*3,cusrc0,cusrc0pos,dt*fesc*factfesc*factgrp[igrp],dx,nsource,aexp,c);
		#else
		      cuAddSource<<<gridsource,blocksource>>>(cuegy_new+igrp*NCELLS3,cuflx_new+igrp*NCELLS3*3,cusrc0,cusrc0pos,dt*fesc*factgrp[igrp],dx,nsource,1.,c);
		#endif
		
		      CUERR();
		      if(verbose) puts("Add Source ok");
		#endif
		
		      if(verbose) puts("Hyperbolic Flux ok");
		
		      hipDeviceSynchronize();

		}

#ifdef TIMINGS     
#ifdef WMPI	  
      mpisynch();
#endif
      if(rank==0)
	{
	  q11=cutGetTimerValue(timer);
	}
#endif
	
#ifdef TESTCOOL  
#ifdef COSMO
      cuComputeIon<<<gridion,blockion>>>(cuegy_new, cuflx_new, cuxion, cudensity, cutemperature, dt/cooling, c, egy_min,unit_number,aexp);
#else
      cuComputeIon<<<gridion,blockion>>>(cuegy_new, cuflx_new, cuxion, cudensity, cutemperature, dt/cooling, c, egy_min,unit_number,1.);
#endif
#endif
      CUERR();
      if(verbose) puts("Chemistry     ok");
      hipDeviceSynchronize();
#ifdef WMPI
      mpisynch();
#endif

#ifdef TIMINGS
      if(rank==0)
	{
	  q4=cutGetTimerValue(timer);
	}
#endif

	  // Here cuegy is used to store the temperature
#ifdef COSMO
      float hubblet=Hubble0*sqrtf(omegam/aexp+omegav*(aexp*aexp))/aexp;
      cuComputeTemp<<<gcool,bcool>>>( cuxion, cudensity, cutemperature, cuegy_new, fudgecool, c, dt/cooling, unit_number, ncvgcool, aexp, hubblet, cuflx_new, clump);
#else
      cuComputeTemp<<<gcool,bcool>>>( cuxion, cudensity, cutemperature, cuegy_new, fudgecool, c, dt/cooling, unit_number, ncvgcool, 1.,   0., cuflx_new, clump);
#endif
      CUERR();
      if(verbose) puts("Cooling  ok");
      hipDeviceSynchronize();
#ifdef WMPI	  
      mpisynch();
#endif

#ifdef TIMINGS
      hipDeviceSynchronize();
#ifdef WMPI
      mpisynch();
#endif
      if(rank==0)
	{
	  q8=cutGetTimerValue(timer);
	}
#endif

      hipMemcpy(cuegy,cuegy_new,NCELLS3*sizeof(float)*NGRP,hipMemcpyDeviceToDevice);
      hipMemcpy(cuflx,cuflx_new,NCELLS3*sizeof(float)*3*NGRP,hipMemcpyDeviceToDevice);


#ifdef TIMINGS
      hipDeviceSynchronize();
#ifdef WMPI
      mpisynch();
#endif
      if(rank==0)
	{
	  q10=cutGetTimerValue(timer);
	}
#endif


      if(verbose) puts("Dealing with boundaries");


#ifndef WMPI
for (int igrp=0;igrp<NGRP;igrp++)
	{
  if(boundary==0) // transmissive boundary conditions
    {
      cusetboundarytrans_xp<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NGRP*3);
      cusetboundarytrans_yp<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NGRP*3);
      cusetboundarytrans_zp<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NGRP*3);
      cusetboundarytrans_xm<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NGRP*3);
      cusetboundarytrans_ym<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NGRP*3);
      cusetboundarytrans_zm<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NGRP*3);
    }
  else if(boundary==1) // reflexive boundary conditions
    {
      cusetboundaryref_zp<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_zm<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_yp<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_ym<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_xp<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_xm<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
    }
  else if(boundary==2) // Periodic boundary conditions
    {
      cusetboundaryper_xp<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_yp<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_zp<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_xm<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_ym<<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryper_zm<<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
    }
  else if(boundary==3) // Mixed boundary conditions
    {
      cusetboundarytrans_xp<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_yp  <<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_zp  <<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundarytrans_xm<<<gridboundx,blockboundx>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_ym  <<<gridboundy,blockboundy>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
      cusetboundaryref_zm  <<<gridboundz,blockboundz>>>(cuegy+igrp*NCELLS3, cuxion, cudensity, cutemperature, cuflx+igrp*NCELLS3*3);
    }
  
}
#else

  
  
  if(neigh[5]!=rank)
    {  
      exchange_zp(cuegy, cuflx, cuegy_new, buff, neigh, pos[2]%2);
      exchange_zm(cuegy, cuflx, cuegy_new, buff, neigh, pos[2]%2);
    }
  else
    {
      cusetboundaryper_zp<<<gridboundz,blockboundz>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      cusetboundaryper_zm<<<gridboundz,blockboundz>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
    }

  if(neigh[3]!=rank)
    {
      exchange_yp(cuegy, cuflx, cuegy_new, buff, neigh, pos[1]%2);
      exchange_ym(cuegy, cuflx, cuegy_new, buff, neigh, pos[1]%2);
    }
  else
    {
      cusetboundaryper_yp<<<gridboundy,blockboundy>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      cusetboundaryper_ym<<<gridboundy,blockboundy>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
    }

  if(neigh[1]!=rank)
    {
      exchange_xp(cuegy, cuflx, cuegy_new, buff, neigh, pos[0]%2);
      exchange_xm(cuegy, cuflx, cuegy_new, buff, neigh, pos[0]%2);
    }
  else
    {
      cusetboundaryper_xp<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      cusetboundaryper_xm<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
    }

  if(boundary==0)
    {
      //printf("coucou\n");
      if(pos[0]==0) cusetboundarytrans_xm<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      if(pos[1]==0) cusetboundarytrans_ym<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      if(pos[2]==0) cusetboundarytrans_zm<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);

      if(pos[0]==(NGPUX-1)) cusetboundarytrans_xp<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      if(pos[1]==(NGPUY-1)) cusetboundarytrans_yp<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);
      if(pos[2]==(NGPUZ-1)) cusetboundarytrans_zp<<<gridboundx,blockboundx>>>(cuegy, cuxion, cudensity, cutemperature, cuflx);

    }

#endif

  hipDeviceSynchronize(); 
#ifdef WMPI
  mpisynch();
#endif

#ifdef TIMINGS
  if(rank==0)
    {
      q7=cutGetTimerValue(timer);
    }
#endif
  
  //printf("proc %d ready to dump\n",ic_rank);

  if(((nstep%ndumps)==0)||(forcedump))
	{
	  ntsteps=ntsteps+1;
	  forcedump=0;
#ifdef COSMO
#ifdef FLAT_COSMO
	  float aexpdump=t2a(t+dt,omegav,Hubble0);
#else
	  if(t+dt>t_int_max)
	    {
	      aexpdump=(a_int[int_step+2]-a_int[int_step+1])/(t_int[int_step+2]-t_int[int_step+1])*(t+dt-t_int[int_step+1]);
	    }
	  else
	    {
	      aexpdump=(a_int[int_step+1]-a_int[int_step])/(t_int[int_step+1]-t_int[int_step])*(t+dt-t_int[int_step]);
	    }
#endif
	  cuDumpResults(ntsteps,t+dt,aexpdump,ic_rank);
#else
	  cuDumpResults(ntsteps,t+dt,0.,ic_rank);
#endif
	  tnext=tnext+ndumps*dt/ft;
	  if(rank==0) printf("tnext=%f\n",tnext/unit_time);
	}

      //--------------------------------------------------------------------
      // Dealing with fieldlists
      //--------------------------------------------------------------------

      ft=fminf(ft*2.,1.);
      
      if(fieldlist)
	{
	  if(changefield)
	    {
	    int ercode;
#ifdef WMPI
	      ercode=cuGetField(ifield,ic_rank);
#else
	      ercode=cuGetField(ifield,0);
#endif
	      if(ercode==38)
		{
		  if(rank==0)
		    {
		      fclose(logfile);
		      fclose(timefile);
		    }
		  abort();
		}
	      forcedump=0;
	      changefield=0;
	      ifield++;
	      tfield=tlist[ifield];
	      ft=1./powf(2.,20);
	      //ft=1.;
	    }
	}


      // UPDATING VARIABLES

      t=t+dt;
      if(t>tmax)
	{
	  puts("t > tmax -----> run will be terminated");
	}
#ifdef COSMO

#ifdef FLAT_COSMO
      aexp=t2a(t,omegav,Hubble0); // A CHANGER PAR INTERPOLATION
#else
      if(t>t_int_max)
	{
	  int_step++;
	}
      aexp=(a_int[int_step+1]-a_int[int_step])/(t_int[int_step+1]-t_int[int_step])*(t-t_int[int_step]);
#endif


      c=c_r/aexp;
#endif       
      
      hipDeviceSynchronize();
#ifdef WMPI
      mpisynch();
#endif
      if(rank==0)
	{
	  q1=cutGetTimerValue(timer);
	}


      nstep++;
      if(nstep==nmax) {
	if(rank==0) puts("Max number of steps achieved: STOP");
	break;
      }

      hipDeviceSynchronize();
#ifdef WMPI
      get_elapsed(&time_new);
      time_new=time_new-time_old;
      mpireducemax(&time_new);
      mpisynch();
#endif

#ifdef TIMINGS
      if(rank==0){
	q9=cutGetTimerValue(timer);
	printf("transport=%f chem=%f cool=%f update=%f bound=%f IO=%f,grand total=%f time_new=%lf\n",q11-q0,q4-q11,q8-q4,q10-q8,q7-q10,q9-q7,q9-q0,time_new);
	fprintf(timefile,"%d %f %f %f %f %f %f %f\n",nstep-1,q11-q0,q4-q11,q8-q4,q10-q8,q7-q10,q9-q7,q9-q0,time_new);
      }


#endif

      hipDeviceSynchronize();
#ifdef WMPI	  
      mpisynch();
#endif

    }

  if(rank==0) fclose(logfile);
#ifdef TIMINGS
  if(rank==0) fclose(timefile);
#endif
  return 0;
}

